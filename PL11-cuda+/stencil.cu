#include "hip/hip_runtime.h"
#include "stencil.h"

#define NUM_BLOCKS 128
#define NUM_THREADS_PER_BLOCK 256
#define SIZE NUM_BLOCKS *NUM_THREADS_PER_BLOCK

using namespace std;

// O id varia entre 0 e o comprimento de a.

__global__ void stencilKernel(float *a, float *c)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int lid = threadIdx.x;                            // local thread id within a block
    __shared__ float temp[NUM_THREADS_PER_BLOCK + 4]; // para o das pontas, porque o temp tem de ter:
    // n1 n2 nMesmo nMesmo ... nMesmo n3 n4
    // Porque para calcular os valores precisas sempre de 2 em cada ponta.
    temp[lid] = a[id];
    if (lid == 1)
    {
        temp[0] = a[NUM_BLOCKS - 2];
        temp[1] = a[NUM_BLOCKS - 1];
    }
    __syncthreads(); // wait for all threads within a block
    // Temos de escrever todos os valores antes de avançar para o cálculo do c
    c[id] = 0;
    for (int n = -2; n <= 2; n++)
    {
        if ((id + n >= 0) && (id + n < SIZE))
            // Tem de ser mais 2, porque o array tmp começa com os 2 primeiros valores das pontas.
            // O lid é necessário para ires buscar ao array temp não as coisas do primeiro bloco, mas as do seguinte.
            c[id] += temp[n + lid + 2];
    }
}

void stencil(float *a, float *c)
{
    chrono::steady_clock::time_point begin = chrono::steady_clock::now();

    for (int i = 0; i < SIZE; i++)
    {
        // considers 4 neighbours
        for (int n = -2; n <= 2; n++)
        {
            if ((i + n >= 0) && (i + n < SIZE))
                c[i] += a[i + n];
        }
    }

    chrono::steady_clock::time_point end = chrono::steady_clock::now();
    cout << endl
         << "Sequential CPU execution: " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << " microseconds" << endl
         << endl;
}

void launchStencilKernel(float *a, float *c)
{
    // pointers to the device memory
    float *da, *dc;
    // declare variable with size of the array in bytes
    int bytes = SIZE * sizeof(float);

    // allocate the memory on the device
    hipMalloc((void **)&da, bytes);
    hipMalloc((void **)&dc, bytes);
    checkCUDAError("mem allocation");

    // copy inputs to the device
    hipMemcpy(&da, a, bytes, hipMemcpyHostToDevice);
    checkCUDAError("memcpy h->d");

    // launch the kernel
    startKernelTime();
    stencilKernel<<<NUM_THREADS_PER_BLOCK, NUM_BLOCKS>>>(da, dc);
    stopKernelTime();
    checkCUDAError("kernel invocation");

    // copy the output to the host
    hipMemcpy((void **)&dc, c, bytes, hipMemcpyDeviceToHost);
    checkCUDAError("memcpy d->h");

    // free the device memory
    hipFree(da);
    hipFree(dc);
    checkCUDAError("mem free");
}

int main(int argc, char **argv)
{
    // arrays on the host
    float a[SIZE], b[SIZE], c[SIZE];

    // initialises the array
    for (unsigned i = 0; i < SIZE; ++i)
        a[i] = (float)rand() / RAND_MAX;

    stencil(a, b);

    launchStencilKernel(a, c);

    return 0;
}